#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../headers/mtx.hpp"

__global__ void sparseMatrixMult(CSRMatrix *A, CSCMatrix *B, CSRMatrix *C);

__global__ void hadamardProduct(int *A, int *B);

__host__ void makeUnit(CSCMatrix *A);

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        std::cout << "Usage: ./main <filename>" << std::endl;
        return 1;
    }

    // variable declaration and memory allocation
    CSRMatrix *adjacent, *p1;
    CSRMatrix *devAdjacent, *devp1;
    CSCMatrix *unitVector;
    CSCMatrix *devUnitVector;
    int blocksPerGrid, threadsPerBlock = 256;

    hipHostMalloc(&p1, sizeof(CSRMatrix));
    hipMalloc(&devAdjacent, sizeof(CSRMatrix));
    hipMalloc(&devp1, sizeof(CSRMatrix));
    hipMalloc(&devUnitVector, sizeof(CSCMatrix));

    // get adjacent matrix and copy to GPU
    CSCMatrix tempAdjacent = readMTX(argv[1]);
    adjacent = new CSRMatrix(tempAdjacent.rows, tempAdjacent.columns, tempAdjacent.nz);
    convert(tempAdjacent, adjacent);
    hipMemcpy(devAdjacent, adjacent, sizeof(CSRMatrix), hipMemcpyHostToDevice);

    // ~~~~~~~ calculate p1
    unitVector = new CSCMatrix(1, adjacent->columns, adjacent->columns);
    makeUnit(unitVector);
    hipMemcpy(devUnitVector, unitVector, sizeof(CSCMatrix), hipMemcpyHostToDevice);
    blocksPerGrid = (adjacent->rows + threadsPerBlock - 1) / threadsPerBlock;
    sparseMatrixMult<<<blocksPerGrid, threadsPerBlock>>>(devAdjacent, devUnitVector, devp1);
    hipMemcpy(p1, devp1, sizeof(CSCMatrix), hipMemcpyDeviceToHost);

    std::cout << "#Rows/Columns: " << adjacent->rows << std::endl;
    std::cout << "#Non-zeros: " << adjacent->nz << std::endl;

    return 0;
}

__global__ void sparseMatrixMult(CSRMatrix *A, CSCMatrix *B, CSRMatrix *C)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < A->rows && col < B->columns)
    {
        int sum = 0;
        for (int i = A->rowIndex[row]; i < A->rowIndex[row + 1]; i++)
        {
            int k = A->nzIndex[i];
            for (int j = B->colIndex[col]; j < B->colIndex[col + 1]; j++)
                if (B->nzIndex[j] == k)
                    sum += A->nzValues[i] * B->nzValues[j];
        }
        if (sum != 0)
        {
            atomicAdd(&C->rowIndex[row + 1], 1);
            int idx = atomicAdd(&C->rowIndex[A->rows], 1);
            C->nzIndex[idx] = col;
            C->nzValues[idx] = sum;
        }
    }

    printf("mpainw\n");
}

__host__ void makeUnit(CSCMatrix *A)
{
    A->colIndex[0] = 0;
    A->colIndex[1] = A->nz;

    for (int i = 0; i < A->nz; i++)
    {
        A->nzIndex[i] = i;
        A->nzValues[i] = 1;
    }
}