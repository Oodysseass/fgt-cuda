#include <iostream>
#include <hip/hip_runtime.h>
#include "../headers/mtx.hpp"


int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        std::cout << "Usage: ./main <filename>" << std::endl;
        return 1;
    }

    // variable declaration and memory allocation
    CSRMatrix *adjacent, *p1;
    CSRMatrix *devAdjacent, *devp1;
    CSCMatrix *unitVector;
    CSCMatrix *devUnitVector;
    int blocksPerGrid, threadsPerBlock = 256;

    hipHostMalloc(&adjacent, sizeof(CSRMatrix));
    hipHostMalloc(&p1, sizeof(CSRMatrix));
    hipHostMalloc(&unitVector, sizeof(CSCMatrix));
    hipMalloc(&devAdjacent, sizeof(CSRMatrix));
    hipMalloc(&devp1, sizeof(CSRMatrix));
    hipMalloc(&devUnitVector, sizeof(CSCMatrix));

    // get adjacent matrix and copy to GPU
    readMTX(adjacent, argv[1]);
    hipMemcpy(devAdjacent, adjacent, sizeof(CSRMatrix), hipMemcpyHostToDevice);

    std::cout << "#Rows/Columns: " << adjacent->rows << std::endl;
    std::cout << "#Non-zeros: " << adjacent->nz << std::endl;

    return 0;
}
