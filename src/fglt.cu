#include "hip/hip_runtime.h"
#include "../headers/fglt.hpp"


__global__ void rawToNet(int *f0, int *f1, int *f2, int *f3, int *f4,
                         int *nf0, int *nf1, int *nf2, int *nf3, int *nf4,
                         int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        nf0[i] = f0[i];
        nf1[i] = f1[i];
        nf2[i] = f2[i] - 2 * f4[i];
        nf3[i] = f3[i] - f4[i];
        nf4[i] = f4[i];
    }
}

__global__ void dZeroOne(int *rows, int *e, int *p1, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        e[i] = 1;
        p1[i] = rows[i + 1] - rows[i];
    }
}

__global__ void dTwoThree(int *rows, int *cols, int *p1, int *p2, int *d3, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        d3[i] = p1[i] * (p1[i] - 1) / 2;
        for (int j = rows[i]; j < rows[i + 1]; j++)
            p2[i] += p1[cols[j]];

        p2[i] -= p1[i];
    }
}

__global__ void dFour(int *rows, int *cols, int *d4, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        // for each non-zero element in A(i, cols[j])
        // calculate the corresponding element in A^2
        for (int j = rows[i]; j < rows[i + 1]; j++)
        {
            int col = cols[j];

            // take advantage of symmetry to use 2 rows
            // instead of row-column
            // to immitate mutiplication
            for (int k = rows[col]; k < rows[col + 1]; k++)
            {
                for (int l = rows[i]; l < rows[i + 1]; l++)
                {
                    // the two rows do not share an element
                    // in this column for sure
                    if (cols[k] < cols[l])
                        break;

                    // all elements are equal to 1
                    // every time there is match in corresponding columns
                    // is a succesful addition to the multiplication
                    if (cols[k] == cols[l])
                    {
                        d4[i]++;
                        break;
                    }
                }
            }
        }
        d4[i] /= 2;
    }
}

__host__ void compute(CSRMatrix *adjacent, int **freq)
{
    // cudafree nothing just for initialization
    CHECK_CUDA(hipFree(0))

    // declare variable
    int *devRowIndex, *devNzIndex;
    int *devf0, *devf1, *devf2, *devf3, *devf4;
    int *devNf0, *devNf1, *devNf2, *devNf3, *devNf4;
    int threadsPerBlock, blocksPerGrid;
    float ms = 0;
    hipEvent_t start, stop, overallStart, overallStop;

    CHECK_CUDA(hipEventCreate(&start))
    CHECK_CUDA(hipEventCreate(&stop))
    CHECK_CUDA(hipEventCreate(&overallStart))
    CHECK_CUDA(hipEventCreate(&overallStop))

    CHECK_CUDA(hipEventRecord(overallStart))
    CHECK_CUDA(hipEventRecord(start))

    // allocate adjacent to device
    CHECK_CUDA(hipMalloc((void **)&devRowIndex,
                          (adjacent->rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNzIndex,
                          (adjacent->nz) * sizeof(int)))

    // copy to device
    CHECK_CUDA(hipMemcpy(devRowIndex, adjacent->rowIndex,
                          (adjacent->rows + 1) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devNzIndex, adjacent->nzIndex,
                          (adjacent->nz) * sizeof(int),
                          hipMemcpyHostToDevice))

    // allocate raw frequencies to device
    CHECK_CUDA(hipMalloc((void **)&devf0, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf1, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf2, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf3, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf4, (adjacent->rows) * sizeof(int)))

    // allocate net frequencies to device
    CHECK_CUDA(hipMalloc((void **)&devNf0, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNf1, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNf2, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNf3, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNf4, (adjacent->rows) * sizeof(int)))

    CHECK_CUDA(hipEventRecord(stop))
    CHECK_CUDA(hipEventSynchronize(stop))
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop))
    printf("Allocations and copy time: %f sec\n", ms);

    // prepare for device functions
    threadsPerBlock = 512;
    blocksPerGrid = (adjacent->rows + threadsPerBlock - 1) / threadsPerBlock;

    CHECK_CUDA(hipEventRecord(start))

    // d0, d1
    std::cout << "Calculate d0, d1" << std::endl;
    dZeroOne<<<blocksPerGrid, threadsPerBlock>>>(devRowIndex, devf0, devf1, adjacent->rows);

    // d1, d2
    std::cout << "Calculate d2, d3" << std::endl;
    dTwoThree<<<blocksPerGrid, threadsPerBlock>>>(devRowIndex, devNzIndex, devf1, devf2, devf3, adjacent->rows);

    // d4
    std::cout << "Calculate d4" << std::endl;
    dFour<<<blocksPerGrid, threadsPerBlock>>>(devRowIndex, devNzIndex, devf4, adjacent->rows);

    // transform to net
    std::cout << "Calculate net frequencies" << std::endl;
    rawToNet<<<blocksPerGrid, threadsPerBlock>>>(devf0, devf1, devf2, devf3, devf4, devNf0, devNf1, devNf2,
                                                 devNf3, devNf4, adjacent->rows);

    CHECK_CUDA(hipEventRecord(stop))
    CHECK_CUDA(hipEventSynchronize(stop))
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop))
    printf("Calculation time: %f sec\n", ms);


    CHECK_CUDA(hipEventRecord(start))

    // copy results to host
    CHECK_CUDA(hipMemcpy(freq[0], devNf0, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[1], devNf1, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[2], devNf2, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[3], devNf3, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[4], devNf4, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));

    // deallocate device memory
    CHECK_CUDA(hipFree(devRowIndex))
    CHECK_CUDA(hipFree(devNzIndex))
    CHECK_CUDA(hipFree(devf0))
    CHECK_CUDA(hipFree(devf1))
    CHECK_CUDA(hipFree(devf2))
    CHECK_CUDA(hipFree(devf3))
    CHECK_CUDA(hipFree(devf4))
    CHECK_CUDA(hipFree(devNf0))
    CHECK_CUDA(hipFree(devNf1))
    CHECK_CUDA(hipFree(devNf2))
    CHECK_CUDA(hipFree(devNf3))
    CHECK_CUDA(hipFree(devNf4))

    CHECK_CUDA(hipEventRecord(stop))
    CHECK_CUDA(hipEventSynchronize(stop))
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop))
    printf("Copy and free time: %.4f sec\n", ms);


    CHECK_CUDA(hipEventRecord(overallStop))
    CHECK_CUDA(hipEventSynchronize(overallStop))
    CHECK_CUDA(hipEventElapsedTime(&ms, overallStart, overallStop))
    printf("Total time elapsed: %.4f sec\n", ms);

    CHECK_CUDA(hipEventDestroy(start))
    CHECK_CUDA(hipEventDestroy(stop))
    CHECK_CUDA(hipEventDestroy(overallStart))
    CHECK_CUDA(hipEventDestroy(overallStop))
}
