#include "hip/hip_runtime.h"
#include "../headers/fglt.hpp"

__global__ void dZeroOne(int *rows, int *e, int *p1, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < N)
    {
        e[i] = 1;
        p1[i] = rows[i + 1] - rows[i];
    }
}

__global__ void dTwoThree(int *rows, int *cols, int *p1, int *p2, int *d3, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < N)
    {
        d3[i] = p1[i] * (p1[i] - 1) / 2;
        for (int j = rows[i]; j < rows[i + 1]; j++)
            p2[i] += p1[cols[j]];

        p2[i] -= p1[i];
    }
}

__global__ void dFour(int *rows, int *cols, int *d4, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < N)
    {
        // for each non-zero element in A(i, cols[j])
        // calculate the corresponding element in A^2
        for (int j = rows[i]; j < rows[i + 1]; j++)
        {
            int col = cols[j];
            
            // take advantage of symmetry to use 2 rows
            // instead of row-column
            // to immitate mutiplication
            for (int k = rows[col]; k < rows[col + 1]; k++)
            {
                for (int l = rows[i]; l < rows[i + 1]; l++)
                {
                    // the two rows do not share an element 
                    // in this column for sure
                    if(cols[k] < cols[l]) break;

                    // all elements are equal to 1
                    // every time there is match in corresponding columns
                    // is a succesful addition to the multiplication
                    if (cols[k] == cols[l])
                    {
                        d4[i]++;
                        break;
                    }
                }
            }
        }
        d4[i] /= 2;
    }
}

__host__ void compute(CSRMatrix *adjacent, int **freq)
{
    // declare variable
    int *devRowIndex, *devNzIndex, *devNzValues;
    int *devf0, *devf1, *devf2, *devf3, *devf4;
    int threadsPerBlock, blocksPerGrid;

    // allocate adjacent to device
    CHECK_CUDA(hipMalloc((void **)&devRowIndex,
                          (adjacent->rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNzIndex,
                          (adjacent->nz) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devNzValues,
                          (adjacent->nz) * sizeof(int)))

    // copy to device
    CHECK_CUDA(hipMemcpy(devRowIndex, adjacent->rowIndex,
                          (adjacent->rows + 1) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devNzIndex, adjacent->nzIndex,
                          (adjacent->nz) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devNzValues, adjacent->nzValues,
                          (adjacent->nz) * sizeof(int),
                          hipMemcpyHostToDevice))

    // allocate frequencies to device
    CHECK_CUDA(hipMalloc((void **)&devf0, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf1, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf2, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf3, (adjacent->rows) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devf4, (adjacent->rows) * sizeof(int)))

    // prepare for device functions
    threadsPerBlock = 512;
    blocksPerGrid = (adjacent->rows + threadsPerBlock - 1) / threadsPerBlock;

    // d0, d1
    std::cout << "Calculate d0, d1" << std::endl;
    dZeroOne<<<blocksPerGrid, threadsPerBlock>>>(devRowIndex, devf0, devf1, adjacent->rows);

    // d1, d2
    std::cout << "Calculate d2, d3" << std::endl;
    dTwoThree<<<blocksPerGrid, threadsPerBlock>>>(devRowIndex, devNzIndex, devf1, devf2, devf3, adjacent->rows);

    // d4
    std::cout << "Calculate d4" << std::endl;
    dFour<<<blocksPerGrid, threadsPerBlock>>>(devRowIndex, devNzIndex, devf4, adjacent->rows);

    // copy results to host
    CHECK_CUDA(hipMemcpy(freq[0], devf0, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[1], devf1, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[2], devf2, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[3], devf3, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(freq[4], devf4, adjacent->rows * sizeof(int), hipMemcpyDeviceToHost));

    // deallocate device memory
    CHECK_CUDA(hipFree(devRowIndex))
    CHECK_CUDA(hipFree(devNzIndex))
    CHECK_CUDA(hipFree(devNzValues))
    CHECK_CUDA(hipFree(devf0))
    CHECK_CUDA(hipFree(devf1))
    CHECK_CUDA(hipFree(devf2))
    CHECK_CUDA(hipFree(devf3))
    CHECK_CUDA(hipFree(devf4))
}
