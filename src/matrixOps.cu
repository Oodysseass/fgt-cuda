#include "hip/hip_runtime.h"
#include "../headers/matrixOps.hpp"

void sparseMult(CSRMatrix *A, CSRMatrix *B, CSRMatrix *C)
{
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    CSRMatrix *devA, *devB, *devC;

    // allocate A
    CHECK_CUDA(hipMalloc((void **)&devA->rowIndex,
                          (A->rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devA->nzIndex,
                          (A->nz) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devA->nzValues,
                          (A->nz) * sizeof(int)))
    // allocate B
    CHECK_CUDA(hipMalloc((void **)&devB->rowIndex,
                          (B->rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devB->nzIndex,
                          (B->nz) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devB->nzValues,
                          (B->nz) * sizeof(int)))
    // allocate only rowIndexes of C
    CHECK_CUDA(hipMalloc((void **)&devC->rowIndex,
                          (A->rows + 1) * sizeof(int)))

    // copy A
    CHECK_CUDA(hipMemcpy(devA->rowIndex, A->rowIndex,
                          (A->rows + 1) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devA->nzIndex, A->nzIndex,
                          (A->nz) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devA->nzValues, A->nzValues,
                          (A->nz) * sizeof(int),
                          hipMemcpyHostToDevice))
    // copy B
    CHECK_CUDA(hipMemcpy(devB->rowIndex, B->rowIndex,
                          (B->rows + 1) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devB->nzIndex, B->nzIndex,
                          (B->nz) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devB->nzValues, B->nzValues,
                          (B->nz) * sizeof(int),
                          hipMemcpyHostToDevice))

    // preparing for cusparse api
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void *dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    // create matrixes in cusparse csr format
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A->rows, A->columns, A->nz,
                                     devA->rowIndex, devA->nzIndex,
                                     devA->nzValues, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I))
    CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B->rows, B->columns, B->nz,
                                     devB->rowIndex, devB->nzIndex,
                                     devB->nzValues, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I))
    CHECK_CUSPARSE(hipsparseCreateCsr(&matC, A->rows, B->columns, 0,
                                     NULL, NULL, NULL, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32I))

    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc))

    float alpha = 1.0f;
    float beta = 0.0f;
    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA,
                                                 matB, &beta, matC, HIP_R_32I,
                                                 HIPSPARSE_SPGEMM_DEFAULT,
                                                 spgemmDesc, &bufferSize1, NULL))
    // check memory requirment of next step
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA,
                                                 matB, &beta, matC, HIP_R_32I,
                                                 HIPSPARSE_SPGEMM_DEFAULT,
                                                 spgemmDesc, &bufferSize1,
                                                 dBuffer1))
    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA,
                                                 matB, &beta, matC, HIP_R_32I,
                                                 HIPSPARSE_SPGEMM_DEFAULT,
                                                 spgemmDesc, &bufferSize2,
                                                 dBuffer2))

    // A * B product
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle, opA, opB, &alpha, matA, matB,
                                          &beta, matC, HIP_R_32I,
                                          HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                          &bufferSize2, dBuffer2))
    // get matrix C non-zero entries C_nnz1
    int64_t tempRows, tempCols, tempnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matC, &tempRows, &tempCols,
                                        &tempnz))
    // allocate matrix C
    CHECK_CUDA(hipMalloc((void **)&devC->nzIndex, tempCols * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&devC->nzValues, tempnz * sizeof(float)))

    // update matC
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matC, devC->rowIndex, devC->nzIndex,
                                          devC->nzValues))
    // copy results
    CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle, opA, opB, &alpha, matA, matB,
                                       &beta, matC, HIP_R_32I, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc))

    // destroy descriptors
    CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
    CHECK_CUSPARSE(hipsparseDestroy(handle))

    // copy to host
    C = new CSRMatrix(tempRows, tempCols, tempnz);
    CHECK_CUDA(hipMemcpy(C->rowIndex, devC->rowIndex, C->rows * sizeof(int),
                          hipMemcpyDeviceToHost))
    CHECK_CUDA(hipMemcpy(C->nzIndex, devC->nzIndex, C->nz * sizeof(int),
                          hipMemcpyDeviceToHost))
    CHECK_CUDA(hipMemcpy(C->nzValues, devC->nzValues, C->nz * sizeof(int),
                          hipMemcpyDeviceToHost))

    CHECK_CUDA(hipFree(dBuffer1))
    CHECK_CUDA(hipFree(dBuffer2))
    CHECK_CUDA(hipFree(devA->rowIndex))
    CHECK_CUDA(hipFree(devA->nzIndex))
    CHECK_CUDA(hipFree(devA->nzValues))
    CHECK_CUDA(hipFree(devB->rowIndex))
    CHECK_CUDA(hipFree(devB->rowIndex))
    CHECK_CUDA(hipFree(devB->nzIndex))
    CHECK_CUDA(hipFree(devC->nzValues))
    CHECK_CUDA(hipFree(devC->nzIndex))
    CHECK_CUDA(hipFree(devC->nzValues))
}

__global__ void calcdZero(int *e, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < N)
        e[i] = 1;
}

__global__ void calcdOne(CSRMatrix *A, int *p1)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < A->rows)
        p1[i] = A->rowIndex[i + 1] - A->rowIndex[i];
}

__global__ void calcdTwo(CSRMatrix *A, int *p1, int *p2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < A->rows)
    {
        for (int j = A->rowIndex[i]; j < A->rowIndex[i + 1]; j++)
            p2[i] += p1[A->nzIndex[j]];

        p2[i] -= p1[i];
    }
}