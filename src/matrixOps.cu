#include "hip/hip_runtime.h"
#include "../headers/matrixOps.hpp"

__global__ void calcdZero(int *e, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < N)
        e[i] = 1;
}

__global__ void calcdOne(CSRMatrix *A, int *p1)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < A->rows)
        p1[i] = A->rowIndex[i + 1] - A->rowIndex[i];
}

__global__ void calcdTwo(CSRMatrix *A, int *p1, int *p2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < A->rows)
    {
        for (int j = A->rowIndex[i]; j < A->rowIndex[i + 1]; j++)
            p2[i] += p1[A->nzIndex[j]];

        p2[i] -= p1[i];
    }
}

__host__ void compute(CSRMatrix *adjacent, int **freq)
{
    // allocate device memory
    CSRMatrix *devAdjacent;
    int **devFreq;

    std::cout << "Allocate devAdjacent" << std::endl;
    CHECK_CUDA(hipMalloc(&devAdjacent, sizeof(CSRMatrix)))
    CHECK_CUDA(hipMalloc(&devAdjacent->rowIndex,
                          (adjacent->rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc(&devAdjacent->nzIndex,
                          (adjacent->nz) * sizeof(int)))
    CHECK_CUDA(hipMalloc(&devAdjacent->nzValues,
                          (adjacent->nz) * sizeof(int)))

    std::cout << "Allocate devFreq" << std::endl;
    CHECK_CUDA(hipMalloc((void **)&devFreq, 5 * sizeof(int *)))
    for (int i = 0; i < 5; i++)
        CHECK_CUDA(hipMalloc((void **)&devFreq[i],
                              (adjacent->rows) * sizeof(int)))

    std::cout << "Copy adjacent" << std::endl;
    // copy to device
    CHECK_CUDA(hipMemcpy(devAdjacent->rowIndex, adjacent->rowIndex,
                          (adjacent->rows + 1) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devAdjacent->nzIndex, adjacent->nzIndex,
                          (adjacent->nz) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(devAdjacent->nzValues, adjacent->nzValues,
                          (adjacent->nz) * sizeof(int),
                          hipMemcpyHostToDevice))

    std::cout << "Free device" << std::endl;
    // deallocate device memory
    CHECK_CUDA(hipFree(devAdjacent->rowIndex))
    CHECK_CUDA(hipFree(devAdjacent->nzIndex))
    CHECK_CUDA(hipFree(devAdjacent->nzValues))

    for (int i = 0; i < 5; i++)
        CHECK_CUDA(hipFree(devFreq[i]))
    CHECK_CUDA(hipFree(devFreq))
    std::cout << "end" << std::endl;
}
