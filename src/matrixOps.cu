#include "matrixOps.hpp"

void sparseMult(CSRMatrix *A, CSRMatrix *B, CSRMatrix *C)
{
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    CSRMatrix *devA, *devB, *devC;

    // allocate A
    CHECK_CUDA( hipMalloc((void **) &devA->rowIndex,
                            (A->rows + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void **) &devA->nzIndex,
                            (A->columns) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void **) &devA->nzValues,
                            (A->nz) * sizeof(int)) );
    // allocate B
    CHECK_CUDA( hipMalloc((void **) &devB->rowIndex,
                            (B->rows + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void **) &devB->nzIndex,
                            (B->columns) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void **) &devB->nzValues,
                            (B->nz) * sizeof(int)) );
    // allocate only rowIndexes of C
    CHECK_CUDA( hipMalloc((void **) &devB->rowIndex,
                            (A->rows) * sizeof(int)) );

    // copy A
    CHECK_CUDA( hipMemcpy(devA->rowIndex, A->rowIndex,
                            (A->rows + 1) * sizeof(int),
                            hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(devA->nzIndex, A->nzIndex,
                            (A->columns) * sizeof(int),
                            hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(devA->nzValues, A->nzValues,
                            (A->nz) * sizeof(int),
                            hipMemcpyHostToDevice) );
    // copy B
    CHECK_CUDA( hipMemcpy(devB->rowIndex, B->rowIndex,
                            (B->rows + 1) * sizeof(int),
                            hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(devB->nzIndex, B->nzIndex,
                            (B->columns) * sizeof(int),
                            hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(devB->nzValues, B->nzValues,
                            (B->nz) * sizeof(int),
                            hipMemcpyHostToDevice) );

}