#include <iostream>
#include <hip/hip_runtime.h>


__global__ void vectorAdd(const float* a, const float* b, float* c, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
        c[i] = a[i] + b[i];
}

int main(int argc, char* argv[])
{
    const int N = 512;
    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    hipHostMalloc(&a, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&b, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&c, N * sizeof(float), hipHostMallocDefault);

    for (int i = 0; i < N; i++)
    {
        a[i] = (float)i;
        b[i] = (float)i * 2;
    }

    hipMalloc(&dev_a, N * sizeof(float));
    hipMalloc(&dev_b, N * sizeof(float));
    hipMalloc(&dev_c, N * sizeof(float));

    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    bool passed = true;
    for (int i = 0; i < N; i++)
    {
        if (c[i] != a[i] + b[i])
        {
            passed = false;
            break;
        }
    }
    std::cout << "Vector addition: " << (passed ? "PASSED" : "FAILED") << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    return 0;
}
